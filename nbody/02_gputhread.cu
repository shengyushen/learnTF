#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */
__global__
void bodyForce_sum0(Body *p,float * px,float* py,float* pz, float dt, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int idx = index; idx < n*n; idx += stride)
  {
		int i=idx/n;
		int j=idx%n;

    float dx = p[j].x - p[i].x;
    float dy = p[j].y - p[i].y;
    float dz = p[j].z - p[i].z;
    float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
    float invDist = rsqrtf(distSqr);
    float invDist3 = invDist * invDist * invDist;
		px[idx]=dx*invDist3;
		py[idx]=dy*invDist3;
		pz[idx]=dz*invDist3;
	}
}

__global__
void bodyForce_sum1(Body *p,float * px,float* py,float* pz, float dt, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(int idx = index; idx < n; idx += stride) {
		for(int i = idx*n+1;i < (idx+1)*n;i++) {
			px[idx*n]+=px[i];
			py[idx*n]+=py[i];
			pz[idx*n]+=pz[i];
		}
	}
}

__global__
void bodyForce_sum2(Body *p,float * px,float* py,float* pz, float dt, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(int idx = index; idx < n; idx += stride) {
		int i=idx;
		p[i].vx += dt*px[i*n];
		p[i].vy += dt*py[i*n];
		p[i].vz += dt*pz[i*n];
	}

}

__global__
void bodyForce_sum3(Body *p,float * px,float* py,float* pz, float dt, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(int idx = index; idx < n; idx += stride) {
		int i=idx;
		p[i].x += p[i].vx*dt;
		p[i].y += p[i].vy*dt;
		p[i].z += p[i].vz*dt;
	}

}

void bodyForce(Body *p,float * px,float *py,float *pz, float dt, int n,int numberOfBlocks,int threadsPerBlock) {
    bodyForce_sum0<<<numberOfBlocks,threadsPerBlock>>>(p,px,py,pz, dt, n); // compute interbody forces
//  for (int i = 0; i < n; ++i) {
//    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
//
//    for (int j = 0; j < n; j++) {
//      float dx = p[j].x - p[i].x;
//      float dy = p[j].y - p[i].y;
//      float dz = p[j].z - p[i].z;
//      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
//      float invDist = rsqrtf(distSqr);
//      float invDist3 = invDist * invDist * invDist;
//
//      px[i*n+j]= dx * invDist3; py[i*n+j] = dy * invDist3; pz[i*n+j] = dz * invDist3;
//		}
//	}

		bodyForce_sum1<<<numberOfBlocks,threadsPerBlock>>>(p,px,py,pz, dt, n); // compute interbody forces

//	for (int i = 0; i < n; ++i) {
//		for (int j = 1; j < n; j++) {
//      px[i*n] += px[i*n+j]; py[i*n] += py[i*n+j]; pz[i*n] += pz[i*n+j];
//    }
//	}



		bodyForce_sum2<<<numberOfBlocks,threadsPerBlock>>>(p,px,py,pz, dt, n); // compute interbody forces
//	for (int i = 0; i < n; ++i) {
//    p[i].vx += dt*px[i*n]; p[i].vy += dt*py[i*n]; p[i].vz += dt*pz[i*n];
//  }


//  for (int i = 0; i < n; ++i) {
//    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
//
//    for (int j = 0; j < n; j++) {
//      float dx = p[j].x - p[i].x;
//      float dy = p[j].y - p[i].y;
//      float dz = p[j].z - p[i].z;
//      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
//      float invDist = rsqrtf(distSqr);
//      float invDist3 = invDist * invDist * invDist;
//
//      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
//    }
//
//    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
//  }
}

int main(const int argc, const char** argv) {

  /*
   * Do not change the value for `nBodies` here. If you would like to modify it,
   * pass values into the command line.
   */

  int nBodies = 2<<11;
  int salt = 0;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

  if (argc > 2) salt = atoi(argv[2]);




	// find out the GPU setting
	  int deviceId;
	  int numberOfSMs;
		hipGetDevice(&deviceId);
		hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	  int threadsPerBlock = 256;
	  int numberOfBlocks = 32 * numberOfSMs;


  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf;
	float *px;
	float *py;
	float *pz;

//  buf = (float *)malloc(bytes);
	hipMallocManaged(&buf,bytes);
	hipMallocManaged(&px,nBodies*nBodies*sizeof(float));
	hipMallocManaged(&py,nBodies*nBodies*sizeof(float));
	hipMallocManaged(&pz,nBodies*nBodies*sizeof(float));

  Body *p = (Body*)buf;

  /*
   * As a constraint of this exercise, `randomizeBodies` must remain a host function.
   */

  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

  double totalTime = 0.0;

  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */

  /*******************************************************************/
  // Do not modify these 2 lines of code.
  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();
  /*******************************************************************/

  /*
   * You will likely wish to refactor the work being done in `bodyForce`,
   * as well as the work to integrate the positions.
   */

    bodyForce(p,px,py,pz, dt, nBodies,numberOfBlocks,threadsPerBlock); // compute interbody forces
		bodyForce_sum3<<<numberOfBlocks,threadsPerBlock>>>(p,px,py,pz, dt, nBodies); // compute interbody forces
	//hipDeviceSynchronize();
  //  for (int i = 0 ; i < nBodies; i++) { // integrate position
  //    p[i].x += p[i].vx*dt;
  //    p[i].y += p[i].vy*dt;
  //    p[i].z += p[i].vz*dt;
  //  }

  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */

  /*******************************************************************/
  // Do not modify the code in this section.
    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }
  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
	hipDeviceSynchronize();

#ifdef ASSESS
  checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
  checkAccuracy(buf, nBodies);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
  salt += 1;
#endif
  /*******************************************************************/

  /*
   * Feel free to modify code below.
   */

//  free(buf);
	hipFree(buf);
	hipFree(px);
	hipFree(py);
	hipFree(pz);
}
